#include "hip/hip_runtime.h"
#include <stdio.h>

#include "fftx3.hpp"
#include "fftx_mddft_gpu_public.h"
#include "fftx_imddft_gpu_public.h"
#include "fftx_mdprdft_gpu_public.h"
#include "fftx_imdprdft_gpu_public.h"
#include "device_macros.h"

#include <stdlib.h>
#include <string.h>

#if defined(FFTX_HIP)
#define GPU_STR "rocfft"
#else
#define GPU_STR "cufft"
#endif

static int M, N, K, K_adj;
static bool writefiles = false;

//  generate file name

static char * generateFileName ( const char *type )
{
	// type is: input ==> random input data; output ==> spiral output data; roc ==> rocFFT output data
	static char fileNameBuff[100];
	sprintf ( fileNameBuff, "mddft3d-%s-%dx%dx%d.dat", type, M, N, K );
	return fileNameBuff;
}

//  write data to file(s) for test repeatability.

static void writeBufferToFile ( const char *type, double *datap )
{
	char *fname = generateFileName ( type );
	FILE *fhandle = fopen ( fname, "w" );
	fprintf ( fhandle, "[ \n" );
	for ( int mm = 0; mm < M; mm++ ) {
		for ( int nn = 0; nn < N; nn++ ) {
			for ( int kk = 0; kk < K; kk++ ) {
				fprintf ( fhandle, "FloatString(\"%.12g\"), FloatString(\"%.12g\"), ", 
						  datap[(kk + nn*K + mm*N*K)*2 + 0], datap[(kk + nn*K + mm*N*K)*2 + 1] );
				if ( kk > 0 && kk % 8 == 0 )
					fprintf ( fhandle, "\n" );
			}
			fprintf ( fhandle, "\n" );
		}
	}
	fprintf ( fhandle, "];\n" );
	
	//  fwrite ( datap, sizeof(double) * 2, M * N * K, fhandle );
	fclose ( fhandle );
	return;
}

static void buildInputBuffer ( double *host_X, double *X, bool genData, bool genComplex, bool useFullK )
{
	int KK = ( useFullK ) ? K : K_adj;
	
	if ( genData ) {					// generate a new data input buffer
		for (int m = 0; m < M; m++) {
			for (int n = 0; n < N; n++) {
				for (int k = 0; k < KK; k++) {
					if ( genComplex ) {
						host_X[(k + n*KK + m*N*KK)*2 + 0] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
						host_X[(k + n*KK + m*N*KK)*2 + 1] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
					}
					else {
						host_X[(k + n*KK + m*N*KK)] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
					}
				}
			}
		}
	}

	int nbytes = M * N * KK * sizeof(double);
	if ( genComplex ) nbytes *= 2;
	DEVICE_MEM_COPY ( X, host_X, nbytes, MEM_COPY_HOST_TO_DEVICE);
	DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );
	return;
}

static void checkOutputBuffers ( double *Y, double *cufft_Y, bool isR2C, bool xfmdir )
{
	int datasz, KK = K;
	bool compCplx = true;				//  compare complex buffers

	if ( isR2C ) {
		//  real (double) to complex
		if ( xfmdir ) {
			//  output buffers are complex, dims M * N * K_adj
			datasz = M * N * K_adj * 2;
			KK = K_adj;
		}
		else {
			//  out buffers are real (double), dims M * N * K
			datasz = M * N * K;
			compCplx = false;
		}
	}
	else {
		//  complex to complex, dims M * N * K
		datasz = M * N * K * 2;
	}

	printf ( "cube = [ %d, %d, %d ]\t%s\t%s\t", M, N, K,
			 ( isR2C ) ? "MDPRDFT" : "MDDFT",
			 ( xfmdir ) ? "(Forward)" : "(Inverse)" );

	double *tmp_Y       = new double [ datasz ];
	double *tmp_cufft_Y = new double [ datasz ];
	DEVICE_MEM_COPY ( tmp_Y,             Y, datasz * sizeof(double), MEM_COPY_DEVICE_TO_HOST );
	DEVICE_MEM_COPY ( tmp_cufft_Y, cufft_Y, datasz * sizeof(double), MEM_COPY_DEVICE_TO_HOST );

	bool correct = true;
	double maxdelta = 0.0;

	for ( int m = 0; m < M; m++ ) {
		for ( int n = 0; n < N; n++ ) {
			for ( int k = 0; k < KK; k++ ) {
				if ( compCplx ) {
					DEVICE_FFT_DOUBLECOMPLEX *host_Y       = (DEVICE_FFT_DOUBLECOMPLEX *) tmp_Y;
					DEVICE_FFT_DOUBLECOMPLEX *host_cufft_Y = (DEVICE_FFT_DOUBLECOMPLEX *) tmp_cufft_Y;
					DEVICE_FFT_DOUBLECOMPLEX s = host_Y      [k + n*KK + m*N*KK];
					DEVICE_FFT_DOUBLECOMPLEX c = host_cufft_Y[k + n*KK + m*N*KK];

					bool elem_correct = ( (abs(s.x - c.x) < 1e-7) && (abs(s.y - c.y) < 1e-7) );
					maxdelta = maxdelta < (double)(abs(s.x -c.x)) ? (double)(abs(s.x -c.x)) : maxdelta ;
					maxdelta = maxdelta < (double)(abs(s.y -c.y)) ? (double)(abs(s.y -c.y)) : maxdelta ;
					correct &= elem_correct;
				}
				else {
					double *host_Y = tmp_Y, *host_cufft_Y = tmp_cufft_Y;
					double deltar = abs ( host_Y[(k + n*KK + m*N*KK)] - host_cufft_Y[(k + n*KK + m*N*KK)] );
					bool   elem_correct = ( deltar < 1e-7 );
					maxdelta = maxdelta < deltar ? deltar : maxdelta ;
					correct &= elem_correct;
				}
			}
		}
	}
	
	printf ( "Correct: %s\tMax delta = %E\t\t##PICKME##\n", (correct ? "True" : "False"), maxdelta );
	fflush ( stdout );

	if ( writefiles ) {
		writeBufferToFile ( (const char *)"spiral-out", (double *)tmp_Y );
		writeBufferToFile ( (const char *)GPU_STR,      (double *)tmp_cufft_Y );
	}
	delete[] tmp_Y;
	delete[] tmp_cufft_Y;

	return;
}

static int NUM_ITERS = 100;

static void	run_transform ( fftx::point_t<3> curr, transformTuple_t *tupl, bool isR2C, bool xfmdir )
{
	DEVICE_EVENT_T start, stop, custart, custop;
	DEVICE_EVENT_CREATE ( &start );
	DEVICE_EVENT_CREATE ( &stop );
	DEVICE_EVENT_CREATE ( &custart );
	DEVICE_EVENT_CREATE ( &custop );

	double *X, *Y;
	double sym[100];  // dummy symbol
	int iters = NUM_ITERS + 10;
	
	M = curr.x[0], N = curr.x[1], K = curr.x[2];
	K_adj = (int) ( K / 2 ) + 1;
	double *host_X;
	DEVICE_FFT_DOUBLEREAL *cufft_Y; 

	if ( isR2C && xfmdir ) {
		//  When is real-2-complex and xfmdir (i.e., forward) input is real (double) of dims M * N * K
		//  and the output array is (complex) of dims M * N * (K/2) + 1)
		DEVICE_MALLOC ( &X,       ( M * N * K     * sizeof(DEVICE_FFT_DOUBLEREAL) ) );
		DEVICE_MALLOC ( &Y,       ( M * N * K_adj * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		DEVICE_MALLOC ( &cufft_Y, ( M * N * K_adj * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		host_X = new double[ M * N * K ];
	}
	else if ( isR2C && !xfmdir ) {
		//  When is real-2-complex and !xfmdir (i.e., inverse) input is complex of dims M * N * (K/2) + 1)
		//  and the output array is (double) of dims M * N * K
		DEVICE_MALLOC ( &X,       ( M * N * K_adj * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		DEVICE_MALLOC ( &Y,       ( M * N * K     * sizeof(DEVICE_FFT_DOUBLEREAL) ) );
		DEVICE_MALLOC ( &cufft_Y, ( M * N * K     * sizeof(DEVICE_FFT_DOUBLEREAL) ) );
		host_X = new double[ M * N * K_adj * 2];
	}
	else {
		// complex-2-complex: input and output are complex of dims M * N * K
		DEVICE_MALLOC ( &X,       ( M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		DEVICE_MALLOC ( &Y,       ( M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		DEVICE_MALLOC ( &cufft_Y, ( M * N * K * sizeof(DEVICE_FFT_DOUBLECOMPLEX) ) );
		host_X = new double[ M * N * K * 2];
	}

	//  want to run and time: 1st iteration; 2nd iteration; then N iterations
	//  Report 1st time, 2nd time, and average of N further iterations
	float *milliseconds   = new float[iters];
	float *cumilliseconds = new float[iters];
	bool check_buff = true;

	DEVICE_FFT_HANDLE plan;
	DEVICE_FFT_RESULT res;
	DEVICE_FFT_TYPE   xfmtype = ( !isR2C ) ? DEVICE_FFT_Z2Z : ( xfmdir ) ? DEVICE_FFT_D2Z : DEVICE_FFT_Z2D ;
	res = DEVICE_FFT_PLAN3D ( &plan, M, N, K, xfmtype );
	if ( res != DEVICE_FFT_SUCCESS ) {
		printf ( "Create DEVICE_FFT_PLAN3D failed with error code %d ... skip buffer check\n", res );
		check_buff = false;
	}

	//  Call the transform init function
	( * tupl->initfp )();
	DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

	// set up data in input buffer: gen data = true,
	// gen complex = true if !isR2C or (isR2C and inverse direction); false otherwise
	// use full K dim = false when (isR2C and inverse direction); true otherwise
	buildInputBuffer ( host_X, X, true,
					   ( !isR2C || ( isR2C && !xfmdir ) ),
					   ! ( isR2C && !xfmdir ) );
	/* if ( writefiles ) { */
	/* 	//  Currently, only for MDDFT, i.e., complex to complex */
	/* 	printf ( "Write input buffer to a file..." ); */
	/* 	writeBufferToFile ( (const char *)"input", host_X ); */
	/* 	printf ( "done\n" ); */
	/* } */

	for ( int ii = 0; ii < iters; ii++ ) {
		//  Call the main transform function
		DEVICE_EVENT_RECORD ( start );
		( * tupl->runfp ) ( Y, X, sym );
		DEVICE_EVENT_RECORD ( stop );
		DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

		DEVICE_EVENT_SYNCHRONIZE ( stop );
		DEVICE_EVENT_ELAPSED_TIME ( &milliseconds[ii], start, stop );

/* #ifdef USE_DIFF_DATA */
/* 		buildInputBuffer ( host_X, X, true, */
/* 						   ( !isR2C || ( isR2C && !xfmdir ) ), */
/* 						   ! ( isR2C && !xfmdir ) ); */
/* #else */
/* 		buildInputBuffer ( host_X, X, false, */
/* 						   ( !isR2C || ( isR2C && !xfmdir ) ), */
/* 						   ! ( isR2C && !xfmdir ) ); */
/* #endif */
	}

	//  Call the destroy function
	( * tupl->destroyfp )();
	DEVICE_CHECK_ERROR ( DEVICE_GET_LAST_ERROR () );

	if ( check_buff ) {
		for ( int ii = 0; ii < iters; ii++ ) {
			DEVICE_EVENT_RECORD ( custart );
			if ( !isR2C ) {
				res = DEVICE_FFT_EXECZ2Z ( plan,
										   (DEVICE_FFT_DOUBLECOMPLEX *) X,
										   (DEVICE_FFT_DOUBLECOMPLEX *) cufft_Y,
										   ( xfmdir ) ? DEVICE_FFT_FORWARD : DEVICE_FFT_INVERSE );
			}
			else {
				if ( xfmdir )
					res = DEVICE_FFT_EXECD2Z ( plan,
											   (DEVICE_FFT_DOUBLEREAL *) X,
											   (DEVICE_FFT_DOUBLECOMPLEX *) cufft_Y );
				else
					res = DEVICE_FFT_EXECZ2D ( plan,
											   (DEVICE_FFT_DOUBLECOMPLEX *) X,
											   (DEVICE_FFT_DOUBLEREAL *) cufft_Y );
			}
			if ( res != DEVICE_FFT_SUCCESS) {
				printf ( "Launch DEVICE_FFT_EXEC failed with error code %d ... skip buffer check\n", res );
				check_buff = false;
				break;
			}
			DEVICE_EVENT_RECORD ( custop );
			DEVICE_EVENT_SYNCHRONIZE ( custop );
			DEVICE_EVENT_ELAPSED_TIME ( &cumilliseconds[ii], custart, custop );

			if ( isR2C && !xfmdir ) {
				//  Input buffer is over-written / corrupted when doing IMDPRDFT  
#ifdef USE_DIFF_DATA
				buildInputBuffer ( host_X, X, true,
								   ( !isR2C || ( isR2C && !xfmdir ) ),
								   ! ( isR2C && !xfmdir ) );
#else
				buildInputBuffer ( host_X, X, false,
								   ( !isR2C || ( isR2C && !xfmdir ) ),
								   ! ( isR2C && !xfmdir ) );
#endif
			}
		}
	}
	DEVICE_SYNCHRONIZE ();

	//  check cufft/rocfft and FFTX got same results
	if ( check_buff ) checkOutputBuffers ( Y, (double *)cufft_Y, isR2C, xfmdir );
	
	//  printf("cube = [ %d, %d, %d ]\t\t ##PICKME## \n", M, N, K);
	printf("%f\tms (SPIRAL) vs\t%f\tms (%s),\t\tFIRST iteration\t##PICKME## \n",
		   milliseconds[0], cumilliseconds[0], GPU_STR);
	printf("%f\tms (SPIRAL) vs\t%f\tms (%s),\t\tSECOND iteration\t##PICKME## \n",
		   milliseconds[1], cumilliseconds[1], GPU_STR);
	
	float cumulSpiral = 0.0, cumulHip = 0.0;
	for ( int ii = 10; ii < iters; ii++ ) {
		cumulSpiral += milliseconds[ii];
		cumulHip    += cumilliseconds[ii];
	} 
	printf("%f\tms (SPIRAL) vs\t%f\tms (%s), AVERAGE over %d iterations (range: 11 - %d) ##PICKME## \n",
		   cumulSpiral / NUM_ITERS, cumulHip / NUM_ITERS, GPU_STR, NUM_ITERS, (10 + NUM_ITERS) );

	DEVICE_FREE ( X );
	DEVICE_FREE ( Y );
	DEVICE_FREE ( cufft_Y );
	delete[] host_X;
	delete[] milliseconds;
	delete[] cumilliseconds;

	return;
}


int main( int argc, char** argv) {

	int iloop = 0;
	bool oneshot = false;
	int iters = NUM_ITERS + 10;

	//  Test is to time on a GPU [CUDA or HIP]
	printf ( "Usage: %s: [ iterations ] [ size: MMxNNxKK ] [ writefiles ]\n", argv[0] );
	if ( argc > 1 ) {
		NUM_ITERS = atoi ( argv[1] );
		iters = NUM_ITERS + 10;
		printf ( "%s: Measure %d iterations, ", argv[0], iters );
		
		if ( argc > 2 ) {
			char * foo = argv[2];
			M = atoi ( foo );
			while ( * foo != 'x' ) foo++;
			foo++ ;
			N = atoi ( foo );
			while ( * foo != 'x' ) foo++;
			foo++ ;
			K = atoi ( foo );
			oneshot = true;
			printf ( "Run size: %dx%dx%d, ", M, N, K );
			
			if ( argc > 3 ) {
				//  Only write files when a specified [single] size is used.  Write data to
				//  files -- spiral input data, spiral output data, and rocFFT/cuFFT output
				writefiles = true;
			}
			printf ( "%s data files\n", (writefiles) ? "WRITE" : "DO NOT write" );
		}
		else {
			printf ( "Run all sizes found in library, " );
			printf ( "%s data files\n", (writefiles) ? "WRITE" : "DO NOT write" );
		}
	}
	else {
		printf ( "%s: Measure %d iterations for all sizes found in the library\n", argv[0], iters );
	}
						  
	fftx::point_t<3> *wcube, curr;

	wcube = fftx_mddft_QuerySizes ();
	if (wcube == NULL) {
		printf ( "%s: Failed to get list of available sizes\n", argv[0] );
		exit (-1);
	}

	if ( oneshot ) {
		for ( iloop = 0; ; iloop++ ) {
			if ( wcube[iloop].x[0] == 0 && wcube[iloop].x[1] == 0 && wcube[iloop].x[2] == 0 ) {
				//  requested size is not in library, print message & exit
				printf ( "%s: Cube { %d, %d, %d } not found in library ... exiting\n", argv[0], M, N, K );
				exit (-1);
			}
			if ( wcube[iloop].x[0] == M && wcube[iloop].x[1] == N && wcube[iloop].x[2] == K ) {
				break;
			}
		}
	}

#if defined(FFTX_HIP)
    //  setup the library
	rocfft_setup();
#endif

	transformTuple_t *tupl, *tupli;
	bool isR2C;

	for ( /* iloop is initialized */ ; ; iloop++ ) {
		curr = wcube[iloop];
		if ( curr.x[0] == 0 && curr.x[1] == 0 && curr.x[2] == 0 ) break;

		printf ( "Cube size { %d, %d, %d } is available\n", curr.x[0], curr.x[1], curr.x[2]);
		tupl  = fftx_mddft_Tuple ( wcube[iloop] );
		tupli = fftx_imddft_Tuple ( wcube[iloop] );
		if ( tupl == NULL || tupli == NULL ) {
			printf ( "Failed to get tuples for cube { %d, %d, %d }\n", curr.x[0], curr.x[1], curr.x[2]);
			continue;
		}

		isR2C = false;			//  do complex-2-complex first
		run_transform ( curr, tupl, isR2C, true );
		run_transform ( curr, tupli, isR2C, false );
		
		tupl  = fftx_mdprdft_Tuple ( wcube[iloop] );
		tupli = fftx_imdprdft_Tuple ( wcube[iloop] );
		if ( tupl == NULL || tupli == NULL ) {
			printf ( "Failed to get tuples for cube { %d, %d, %d }\n", curr.x[0], curr.x[1], curr.x[2]);
			continue;
		}

		isR2C = true;			//  do R2c & C2R
		run_transform ( curr, tupl, isR2C, true );
		run_transform ( curr, tupli, isR2C, false );
		
		if ( oneshot ) break;
	}

#if defined(FFTX_HIP)
	//  cleanup the library
	rocfft_cleanup();
#endif

}

