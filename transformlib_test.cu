#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime_api.h>

#include "fftx3.hpp"
#include "fftx_mddft_public.h"

static int M, N, K;

static void buildInputBuffer(double *host_X, double *X)
{
	for (int m = 0; m < M; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				host_X[(k + n*K + m*N*K)*2 + 0] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
				host_X[(k + n*K + m*N*K)*2 + 1] = 1 - ((double) rand()) / (double) (RAND_MAX/2);
			}
		}
	}

	hipMemcpy(X, host_X, M*N*K*2*sizeof(double), hipMemcpyHostToDevice);
	return;
}

static void checkOutputBuffers ( double *Y, double *cufft_Y )
{
	printf("cube = [ %d, %d, %d ]\t", M, N, K);
	hipfftDoubleComplex *host_Y       = new hipfftDoubleComplex[M*N*K];
	hipfftDoubleComplex *host_cufft_Y = new hipfftDoubleComplex[M*N*K];

	hipMemcpy(host_Y      ,       Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);
	hipMemcpy(host_cufft_Y, cufft_Y, M*N*K*sizeof(hipfftDoubleComplex), hipMemcpyDeviceToHost);

	bool correct = true;
	int errCount = 0;
	double maxdelta = 0.0;

	for (int m = 0; m < 1; m++) {
		for (int n = 0; n < N; n++) {
			for (int k = 0; k < K; k++) {
				hipfftDoubleComplex s = host_Y      [k + n*K + m*N*K];
				hipfftDoubleComplex c = host_cufft_Y[k + n*K + m*N*K];
	    
				bool elem_correct =
					(abs(s.x - c.x) < 1e-7) &&
					(abs(s.y - c.y) < 1e-7);
				maxdelta = maxdelta < (double)(abs(s.x -c.x)) ? (double)(abs(s.x -c.x)) : maxdelta ;
				maxdelta = maxdelta < (double)(abs(s.y -c.y)) ? (double)(abs(s.y -c.y)) : maxdelta ;

				correct &= elem_correct;
				if (!elem_correct && errCount < 10) 
				{
					correct = false;
					errCount++;
					//  printf("error at (%d,%d,%d): %f+%fi instead of %f+%fi\n", k, n, m, s.x, s.y, c.x, c.y);
				}
			}
		}
	}
	
	printf ( "Correct: %s\tMax delta = %E\t\t##PICKME## \n", (correct ? "True" : "False"), maxdelta );
	fflush ( stdout );
	delete[] host_Y;
	delete[] host_cufft_Y;

	return;
}

int main() {

	fftx::point_t<3> *wcube, curr;
	int iloop = 0;
	double *X, *Y;
	double sym[100];  // dummy symbol
						  
	//  hipEvent_t start, stop, custart, custop;

	wcube = fftx_mddft_QuerySizes ();
	if (wcube == NULL) {
		printf ( "Failed to get list of available sizes\n" );
		exit (-1);
	}

	transformTuple_t *tupl;
	for ( iloop = 0; ; iloop++ ) {
		curr = wcube[iloop];
		if ( curr.x[0] == 0 && curr.x[1] == 0 && curr.x[2] == 0 ) break;

		printf ( "Cube size { %d, %d, %d } is available\n", curr.x[0], curr.x[1], curr.x[2]);
		tupl = fftx_mddft_Tuple ( wcube[iloop] );
		if ( tupl == NULL ) {
			printf ( "Failed to get tuple for cube { %d, %d, %d }\n", curr.x[0], curr.x[1], curr.x[2]);
		}
		else {
			M = curr.x[0], N = curr.x[1], K = curr.x[2];
			printf ( "M = %d, N = %d, K = %d, malloc sizes = %d * sizeof(double)\n", M, N, K, M*N*K*2 );
		
			hipMalloc(&X,M*N*K*2*sizeof(double));
			hipMalloc(&Y,M*N*K*2*sizeof(double));

			double *host_X = new double[M*N*K*2];

			hipfftDoubleComplex *cufft_Y; 
			hipMalloc(&cufft_Y, M*N*K * sizeof(hipfftDoubleComplex));

			hipfftHandle plan;
			if (hipfftPlan3d(&plan, M, N, K,  HIPFFT_Z2Z) != HIPFFT_SUCCESS) {
				exit(-1);
			}

			//  Call the init function
			( * tupl->initfp )();
			checkCudaErrors ( hipGetLastError () );
 
			// set up data in input buffer and run the transform
			buildInputBuffer(host_X, X);

			for ( int kk = 0; kk < 100; kk++ ) {
				//  try the run function

				( * tupl->runfp ) ( Y, X, sym );
				checkCudaErrors ( hipGetLastError () );
			}
			
			// Tear down / cleanup
			( * tupl->destroyfp ) ();				//  destroy_mddft3d();
			checkCudaErrors ( hipGetLastError () );

			if (hipfftExecZ2Z(
					plan,
					(hipfftDoubleComplex *) X,
					(hipfftDoubleComplex *) cufft_Y,
					HIPFFT_FORWARD
					) != HIPFFT_SUCCESS) {
				printf("hipfftExecZ2Z launch failed\n");
				exit(-1);
			}

			hipDeviceSynchronize();
			if (hipGetLastError() != hipSuccess) {
				printf("hipfftExecZ2Z failed\n");
				exit(-1);
			}

			//  check cufft and CUDA got same results
			checkOutputBuffers ( Y, (double *)cufft_Y );
			
			hipFree ( X );
			hipFree ( Y );
			hipFree ( cufft_Y );
			delete[] host_X;
		}
	}

}
